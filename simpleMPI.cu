#include "hip/hip_runtime.h"
/* Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* Simple example demonstrating how to use MPI with CUDA
*
*  Generate some random numbers on one node.
*  Dispatch them to all nodes.
*  Compute their square root on each node's GPU.
*  Compute the average of the results using MPI.
*
*  simpleMPI.cu: GPU part, compiled with nvcc
*/

#include <iostream>
using std::cerr;
using std::endl;

#include "simpleMPI.h"

// Error handling macro
#define CUDA_CHECK(call)                                                 \
  if ((call) != hipSuccess) {                                           \
    hipError_t err = hipGetLastError();                                \
    cerr << "CUDA error calling \"" #call "\", code is " << err << endl; \
    my_abort(err);                                                       \
  }

// Device code
// Very simple GPU Kernel that computes square roots of input numbers
__global__ void simpleMPIKernel(float *input, float *output) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  output[tid] = sqrt(input[tid]);
}

// Initialize an array with random data (between 0 and 1)
void initData(float *data, int dataSize) {
  for (int i = 0; i < dataSize; i++) {
    data[i] = (float)rand() / RAND_MAX;
  }
}

// CUDA computation on each node
// No MPI here, only CUDA
void computeGPU(float *hostData_out, float *hostData_in, int blockSize, int gridSize, int commRank) {
  int dataSize = blockSize * gridSize;

  // Create CUDA event
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));

  // Allocate data on GPU memory
  float *deviceInputData = NULL;
  CUDA_CHECK(hipMalloc((void **)&deviceInputData, dataSize * sizeof(float)));

  float *deviceOutputData = NULL;
  CUDA_CHECK(hipMalloc((void **)&deviceOutputData, dataSize * sizeof(float)));

  // Record GPU start time
  CUDA_CHECK(hipEventRecord(start));

  // Copy to GPU memory
  CUDA_CHECK(hipMemcpy(deviceInputData, hostData_in, dataSize * sizeof(float),
                        hipMemcpyHostToDevice));

  // Run kernel
  simpleMPIKernel<<<gridSize, blockSize>>>(deviceInputData, deviceOutputData);

  // Copy data back to CPU memory
  CUDA_CHECK(hipMemcpy(hostData_out, deviceOutputData, dataSize * sizeof(float),
                        hipMemcpyDeviceToHost));

  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));

  float elapsed_time_in_ms;
  hipEventElapsedTime(&elapsed_time_in_ms, start, stop);
  std::cout << "[" << commRank << "] Elapsed Time: " << elapsed_time_in_ms << " ms." << std::endl; 

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));
  
  // Free GPU memory
  CUDA_CHECK(hipFree(deviceInputData));
  CUDA_CHECK(hipFree(deviceOutputData));
}

void computeGPU_zerocopy(float *hostOutputData, float *hostInputData, int blockSize, int gridSize, int commRank) {
  // Create CUDA event
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));

  // Record GPU start time
  CUDA_CHECK(hipEventRecord(start));

  // Allocate data on GPU memory
  float *deviceInputData = NULL;
  CUDA_CHECK(hipHostGetDevicePointer((void **)&deviceInputData,  (void *)hostInputData, 0));
  
  float *deviceOutputData = NULL;
  CUDA_CHECK(hipHostGetDevicePointer((void **)&deviceOutputData, (void *)hostOutputData, 0));

  // Run kernel
  simpleMPIKernel<<<gridSize, blockSize>>>(deviceInputData, deviceOutputData);

  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));

  float elapsed_time_in_ms;
  hipEventElapsedTime(&elapsed_time_in_ms, start, stop);
  std::cout << "[" << commRank << "] Elapsed Time: " << elapsed_time_in_ms << " ms." << std::endl; 

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));
}

float sum(float *data, int size) {
  float accum = 0.f;

  for (int i = 0; i < size; i++) {
    accum += data[i];
  }

  return accum;
}
